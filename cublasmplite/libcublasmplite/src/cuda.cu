#include "hip/hip_runtime.h"

#include <nvshmem.h>

#include <memory>
#include <cstdio>
#include <hipblas.h>
#include <iostream>
#include <cstdint>
#include <hip/hip_fp8.h>
#include <hip/hip_fp16.h>

#include "cublasmplite.h"

#include "macros.hpp.inc"

using namespace hipcublasmplite;

// stream_t

stream_t::stream_t() {
    CUBLASMPLITE_CUDA_CHECK(hipStreamCreate(&stream));
    alive = true;
}

stream_t::~stream_t() {
    if(alive) {
        CUBLASMPLITE_CUDA_CHECK(hipStreamDestroy(stream));
    }
}

void stream_t::synchronize() const {
    CUBLASMPLITE_ASSERT(alive);
    CUBLASMPLITE_CUDA_CHECK(hipStreamSynchronize(stream));
}

stream_t::stream_t(stream_t&& that) {
    stream = that.stream;
    alive = that.alive;
    that.stream = (hipStream_t)nullptr;
    that.alive = false;
}

stream_t& stream_t::operator=(stream_t&& that) {
    std::swap(stream, that.stream);
    std::swap(alive, that.alive);
    return *this;
}

stream_t::operator hipStream_t() const { 
    CUBLASMPLITE_ASSERT(alive);
    return stream;
}

hipStream_t stream_t::handle() const { 
    CUBLASMPLITE_ASSERT(alive);
    return stream;
}

void stream_t::wait(hipEvent_t event) const {
    CUBLASMPLITE_ASSERT(alive);
    CUBLASMPLITE_CUDA_CHECK(hipStreamWaitEvent(stream, event));
}

// event_t

event_t::event_t() {
    CUBLASMPLITE_CUDA_CHECK(hipEventCreate(&event));
    alive = true;
}

event_t::~event_t() {
    if(alive) {
        CUBLASMPLITE_CUDA_CHECK(hipEventDestroy(event));
    }
}

event_t::event_t(event_t&& that) {
    event = that.event;
    alive = that.alive;
    that.event = (hipEvent_t)nullptr;
    that.alive = false;
}

event_t& event_t::operator=(event_t&& that) {
    std::swap(event, that.event);
    std::swap(alive, that.alive);
    return *this;
}

event_t::operator hipEvent_t() const { 
    CUBLASMPLITE_ASSERT(alive);
    return event;
}

hipEvent_t event_t::handle() const { 
    CUBLASMPLITE_ASSERT(alive);
    return event;
}

void event_t::record(hipStream_t stream) const {
    CUBLASMPLITE_ASSERT(alive);
    CUBLASMPLITE_CUDA_CHECK(hipEventRecord(event, stream));
}

float event_t::elapsed_time_ms(hipEvent_t stop) const {
    CUBLASMPLITE_ASSERT(alive);
    float time_ms = 0;
    CUBLASMPLITE_CUDA_CHECK(hipEventElapsedTime(&time_ms, event, stop));
    return time_ms;
}

// nvshmem_vector_t

template<typename T> 
nvshmem_vector_t<T>::nvshmem_vector_t(size_t size) : device_vector_view_t<T>(nullptr, size) {
    if (this->_size == 0) {
        this->_ptr_d = nullptr;
    } else {
        this->_ptr_d = (T*)nvshmem_malloc(sizeof(T) * size);
        CUBLASMPLITE_ASSERT(this->_ptr_d != nullptr);
    }
}

template<typename T> 
nvshmem_vector_t<T>::nvshmem_vector_t(size_t size, const T& value) : device_vector_view_t<T>(nullptr, size) {
    if (this->_size == 0) {
        this->_ptr_d = nullptr;
    } else {
        this->_ptr_d = (T*)nvshmem_malloc(sizeof(T) * size);
        CUBLASMPLITE_ASSERT(this->_ptr_d != nullptr);
        std::vector<T> data(size, value);
        CUBLASMPLITE_CUDA_CHECK(hipMemcpy(this->_ptr_d, data.data(), data.size() * sizeof(T), hipMemcpyDefault));
    }
}

template<typename T> 
nvshmem_vector_t<T>::nvshmem_vector_t(const std::vector<T>& data) : device_vector_view_t<T>(nullptr, data.size()) {
    if (data.size() == 0) {
        this->_ptr_d = nullptr;
    } else {
        this->_ptr_d = (T*)nvshmem_malloc(sizeof(T) * data.size());
        CUBLASMPLITE_ASSERT(this->_ptr_d != nullptr);
        CUBLASMPLITE_CUDA_CHECK(hipMemcpy(this->_ptr_d, data.data(), data.size() * sizeof(T), hipMemcpyDefault));
    }
}

template<typename T> 
nvshmem_vector_t<T>::~nvshmem_vector_t() {
    if(this->_ptr_d != nullptr) {
        nvshmem_free(this->_ptr_d);
    }
}

// device_vector_const_view_t

template<typename T>
device_vector_const_view_t<T>::device_vector_const_view_t(device_vector_const_view_t<T>&& that) {
    this->_size = that._size;
    this->_ptr_d = that._ptr_d;
    that._size = 0;
    that._ptr_d = nullptr;
}

template<typename T>
device_vector_const_view_t<T>& device_vector_const_view_t<T>::operator=(device_vector_const_view_t<T>&& that) {
    std::swap(this->_size, that._size);
    std::swap(this->_ptr_d, that._ptr_d);
    return *this;
}

template<typename T>
device_vector_const_view_t<T>::operator std::vector<T>() {
    std::vector<T> out(_size);
    CUBLASMPLITE_CUDA_CHECK(hipMemcpy(out.data(), this->_ptr_d, this->_size * sizeof(T), hipMemcpyDefault));
    return out;
}

// device_vector_view_t

template<typename T>
device_vector_view_t<T>::device_vector_view_t(device_vector_view_t<T>&& that) {
    this->_size = that._size;
    this->_ptr_d = that._ptr_d;
    that._size = 0;
    that._ptr_d = nullptr;
}

template<typename T>
device_vector_view_t<T>& device_vector_view_t<T>::operator=(device_vector_view_t<T>&& that) {
    std::swap(this->_size, that._size);
    std::swap(this->_ptr_d, that._ptr_d);
    return *this;
}

template<typename T>
device_vector_view_t<T>::operator std::vector<T>() {
    return (std::vector<T>) device_vector_const_view_t(this->_ptr_d, this->_size);
}

// device_vector_t

template<typename T>
device_vector_t<T>::device_vector_t(size_t size) : device_vector_view_t<T>(nullptr, size) {
    if (size == 0) {
        this->_ptr_d = nullptr;
    } else {
        CUBLASMPLITE_CUDA_CHECK(hipMalloc(&this->_ptr_d, size * sizeof(T)));
        CUBLASMPLITE_CUDA_CHECK(hipMemset(this->_ptr_d, 0, size * sizeof(T)));
    }
}

template<typename T>
device_vector_t<T>::device_vector_t(const std::vector<T>& data) : device_vector_view_t<T>(nullptr, data.size()) {
    if (data.size() == 0) {
        this->_ptr_d = nullptr;
    } else {
        CUBLASMPLITE_CUDA_CHECK(hipMalloc(&this->_ptr_d, data.size() * sizeof(T)));
        CUBLASMPLITE_CUDA_CHECK(hipMemcpy(this->_ptr_d, data.data(), data.size() * sizeof(T), hipMemcpyDefault));
    }
}

template<typename T>
device_vector_t<T>::~device_vector_t() {
    if (this->_ptr_d != nullptr) {
        CUBLASMPLITE_CUDA_CHECK(hipFree(this->_ptr_d));
    }
}

//////// TODO: fix this madness

template nvshmem_vector_t<hip_bfloat16>::nvshmem_vector_t(size_t size);
template nvshmem_vector_t<hip_bfloat16>::nvshmem_vector_t(size_t size, const hip_bfloat16&);
template nvshmem_vector_t<hip_bfloat16>::nvshmem_vector_t(const std::vector<hip_bfloat16>& data);
template nvshmem_vector_t<hip_bfloat16>::~nvshmem_vector_t();

template nvshmem_vector_t<uint64_t>::nvshmem_vector_t(size_t size);
template nvshmem_vector_t<uint64_t>::nvshmem_vector_t(size_t size, const uint64_t&);
template nvshmem_vector_t<uint64_t>::nvshmem_vector_t(const std::vector<uint64_t>& data);
template nvshmem_vector_t<uint64_t>::~nvshmem_vector_t();

template nvshmem_vector_t<char>::nvshmem_vector_t(size_t size);
template nvshmem_vector_t<char>::nvshmem_vector_t(size_t size, const char&);
template nvshmem_vector_t<char>::nvshmem_vector_t(const std::vector<char>& data);
template nvshmem_vector_t<char>::~nvshmem_vector_t();

template device_vector_const_view_t<hip_bfloat16>::device_vector_const_view_t(device_vector_const_view_t<hip_bfloat16>&&);
template device_vector_const_view_t<hip_bfloat16>& device_vector_const_view_t<hip_bfloat16>::operator=(device_vector_const_view_t<hip_bfloat16>&&);
template device_vector_const_view_t<hip_bfloat16>::operator std::vector<hip_bfloat16>();

template device_vector_const_view_t<__hip_fp8_e4m3_fnuz>::device_vector_const_view_t(device_vector_const_view_t<__hip_fp8_e4m3_fnuz>&&);
template device_vector_const_view_t<__hip_fp8_e4m3_fnuz>& device_vector_const_view_t<__hip_fp8_e4m3_fnuz>::operator=(device_vector_const_view_t<__hip_fp8_e4m3_fnuz>&&);
template device_vector_const_view_t<__hip_fp8_e4m3_fnuz>::operator std::vector<__hip_fp8_e4m3_fnuz>();

template device_vector_const_view_t<__hip_fp8_e5m2_fnuz>::device_vector_const_view_t(device_vector_const_view_t<__hip_fp8_e5m2_fnuz>&&);
template device_vector_const_view_t<__hip_fp8_e5m2_fnuz>& device_vector_const_view_t<__hip_fp8_e5m2_fnuz>::operator=(device_vector_const_view_t<__hip_fp8_e5m2_fnuz>&&);
template device_vector_const_view_t<__hip_fp8_e5m2_fnuz>::operator std::vector<__hip_fp8_e5m2_fnuz>();

template device_vector_const_view_t<char>::device_vector_const_view_t(device_vector_const_view_t<char>&&);
template device_vector_const_view_t<char>& device_vector_const_view_t<char>::operator=(device_vector_const_view_t<char>&&);
template device_vector_const_view_t<char>::operator std::vector<char>();

template device_vector_const_view_t<uint64_t>::device_vector_const_view_t(device_vector_const_view_t<uint64_t>&&);
template device_vector_const_view_t<uint64_t>& device_vector_const_view_t<uint64_t>::operator=(device_vector_const_view_t<uint64_t>&&);
template device_vector_const_view_t<uint64_t>::operator std::vector<uint64_t>();

template device_vector_view_t<hip_bfloat16>::operator std::vector<hip_bfloat16>();
template device_vector_view_t<hip_bfloat16>::device_vector_view_t(device_vector_view_t<hip_bfloat16>&&);
template device_vector_view_t<hip_bfloat16>& device_vector_view_t<hip_bfloat16>::operator=(device_vector_view_t<hip_bfloat16>&&);

template device_vector_view_t<char>::operator std::vector<char>();
template device_vector_view_t<char>::device_vector_view_t(device_vector_view_t<char>&&);
template device_vector_view_t<char>& device_vector_view_t<char>::operator=(device_vector_view_t<char>&&);

template device_vector_view_t<int32_t>::operator std::vector<int32_t>();
template device_vector_view_t<int32_t>::device_vector_view_t(device_vector_view_t<int32_t>&&);
template device_vector_view_t<int32_t>& device_vector_view_t<int32_t>::operator=(device_vector_view_t<int32_t>&&);

template device_vector_view_t<uint64_t>::operator std::vector<uint64_t>();
template device_vector_view_t<uint64_t>::device_vector_view_t(device_vector_view_t<uint64_t>&&);
template device_vector_view_t<uint64_t>& device_vector_view_t<uint64_t>::operator=(device_vector_view_t<uint64_t>&&);

template device_vector_t<hip_bfloat16>::device_vector_t(size_t size);
template device_vector_t<hip_bfloat16>::device_vector_t(const std::vector<hip_bfloat16>& data);
template device_vector_t<hip_bfloat16>::~device_vector_t();

template device_vector_t<int32_t>::device_vector_t(size_t size);
template device_vector_t<int32_t>::device_vector_t(const std::vector<int32_t>& data);
template device_vector_t<int32_t>::~device_vector_t();

template device_vector_t<char>::device_vector_t(size_t size);
template device_vector_t<char>::device_vector_t(const std::vector<char>& data);
template device_vector_t<char>::~device_vector_t();

template device_vector_t<__hip_fp8_e4m3_fnuz>::device_vector_t(size_t size);
template device_vector_t<__hip_fp8_e4m3_fnuz>::device_vector_t(const std::vector<__hip_fp8_e4m3_fnuz>& data);
template device_vector_t<__hip_fp8_e4m3_fnuz>::~device_vector_t();

template device_vector_t<__hip_fp8_e5m2_fnuz>::device_vector_t(size_t size);
template device_vector_t<__hip_fp8_e5m2_fnuz>::device_vector_t(const std::vector<__hip_fp8_e5m2_fnuz>& data);
template device_vector_t<__hip_fp8_e5m2_fnuz>::~device_vector_t();

