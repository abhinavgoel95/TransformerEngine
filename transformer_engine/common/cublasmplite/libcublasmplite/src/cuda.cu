
#include <nvshmem.h>

#include <memory>
#include <cstdio>
#include <hipblas.h>
#include <iostream>
#include <cstdint>
#include <hip/hip_fp8.h>
#include <hip/hip_fp16.h>

#include "cublasmplite.h"

#include "macros.hpp.inc"

using namespace hipcublasmplite;

// stream_t

stream_t::stream_t() {
    CUBLASMPLITE_CUDA_CHECK(hipStreamCreate(&stream));
    alive = true;
}

stream_t::~stream_t() {
    if(alive) {
        CUBLASMPLITE_CUDA_CHECK(hipStreamDestroy(stream));
    }
}

void stream_t::synchronize() const {
    CUBLASMPLITE_ASSERT(alive);
    CUBLASMPLITE_CUDA_CHECK(hipStreamSynchronize(stream));
}

stream_t::stream_t(stream_t&& that) {
    stream = that.stream;
    alive = that.alive;
    that.stream = (hipStream_t)nullptr;
    that.alive = false;
}

stream_t& stream_t::operator=(stream_t&& that) {
    std::swap(stream, that.stream);
    std::swap(alive, that.alive);
    return *this;
}

stream_t::operator hipStream_t() const { 
    CUBLASMPLITE_ASSERT(alive);
    return stream;
}

hipStream_t stream_t::handle() const { 
    CUBLASMPLITE_ASSERT(alive);
    return stream;
}

void stream_t::wait(hipEvent_t event) const {
    CUBLASMPLITE_ASSERT(alive);
    CUBLASMPLITE_CUDA_CHECK(hipStreamWaitEvent(stream, event));
}

// event_t

event_t::event_t() {
    CUBLASMPLITE_CUDA_CHECK(hipEventCreate(&event));
    alive = true;
}

event_t::~event_t() {
    if(alive) {
        CUBLASMPLITE_CUDA_CHECK(hipEventDestroy(event));
    }
}

event_t::event_t(event_t&& that) {
    event = that.event;
    alive = that.alive;
    that.event = (hipEvent_t)nullptr;
    that.alive = false;
}

event_t& event_t::operator=(event_t&& that) {
    std::swap(event, that.event);
    std::swap(alive, that.alive);
    return *this;
}

event_t::operator hipEvent_t() const { 
    CUBLASMPLITE_ASSERT(alive);
    return event;
}

hipEvent_t event_t::handle() const { 
    CUBLASMPLITE_ASSERT(alive);
    return event;
}

void event_t::record(hipStream_t stream) const {
    CUBLASMPLITE_ASSERT(alive);
    CUBLASMPLITE_CUDA_CHECK(hipEventRecord(event, stream));
}

float event_t::elapsed_time_ms(hipEvent_t stop) const {
    CUBLASMPLITE_ASSERT(alive);
    float time_ms = 0;
    CUBLASMPLITE_CUDA_CHECK(hipEventElapsedTime(&time_ms, event, stop));
    return time_ms;
}
