#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <hip/hip_runtime.h>
#include <cudaTypedefs.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cfloat>
#include <cuda/barrier>
#include <utility>

#include "common/common.h"
#include "common/recipe/recipe_common.cuh"
#include "common/transpose/cast_transpose.h"
#include "common/utils.cuh"
#include "nvfp4_utils.cuh"

namespace quantize_transpose_nvfp4 {
namespace {

template <typename T>
constexpr T RoundUpDivide(const T x, const T y) {
  static_assert(std::is_integral<T>::value, "RoundUpDivide requires integral types.");
  static_assert(std::is_unsigned<T>::value, "RoundUpDivide requires unsigned types.");
  return (x + (y - 1)) / y;
}

constexpr int kThreadsPerWarp = 32;

#if CUDA_VERSION >= 12080
// for fp4, we use uint8_t to store 2 fp4 numbers
constexpr int kNFP4PerContainer = 2;

// Hyperparameters for performance tuning
constexpr int kTileDim = 128;
// constexpr int kScaleDim = 32;
constexpr int kNVecIn = 8;             // The number of elements each LDG touches
constexpr int kNVecOut = 16;           // The number of elements each STG touches
constexpr int kNVecSMem = 2;           // The number of elements each LDS/STS touches
constexpr int kThreadsPerBlock = 256;  // Thread block size, 8 warps in total

constexpr int kNVecContainer = kNVecOut / kNFP4PerContainer;

// Auto-calculated constants, do not modify directly)
static_assert(kNVecIn % kNVecSMem == 0, "kNVecIn must be divisible by kNVecSMem");
static_assert(kNVecOut % kNVecSMem == 0, "kNVecOut must be divisible by kNVecSMem");
constexpr int kSMemRow = kTileDim;
constexpr int kSMemCol = (kTileDim / kNVecSMem) + 1;
constexpr int kSMemSize = kSMemRow * kSMemCol * kNVecSMem;
constexpr int kNumThreadsLoad = kTileDim / kNVecIn;
constexpr int kNumThreadsStore = kTileDim / kNVecOut;
// constexpr int kNumThreadsReduce = kScaleDim / kNVecOut;
static_assert(kNumThreadsLoad <= kThreadsPerWarp, "kNumThreadsLoad must be <= kThreadsPerWarp");
static_assert(kNumThreadsStore <= kThreadsPerWarp, "kNumThreadsStore must be <= kThreadsPerWarp");

template <class ScaleType>
__device__ __forceinline__ size_t scale_factor_swizzled_offset(size_t row_idx, size_t col_idx,
                                                               uint32_t col_length) {
  // This function takes in indices from the scale factor matrix and returns an offset in the
  // swizzled format. row_idx, col_idx are original indices from the scale factor matrix (unswizzled
  // index). col_length is the column length of the scale factor matrix. tile_scales_inv is the
  // pointer to the scale factor matrix.

  // https://github.com/NVIDIA/cutlass/blob/main/media/docs/cpp/blackwell_functionality.md#scale-factor-layouts
  // For any scale factor matrix, it's 512B base block. Each base block consists of 128 rows and 4
  // columns. Base block is divided into 4 column blocks, each column block has 32 rows and 4
  // columns.

  // NOTE: There are not a lot of good illustrations about the swizzled scale factor matrix.
  // To think in high level, the swizzled scale factor matrix could be composed as:
  // unswizzled_scale_factor_matrix = torch.empty((M, N // 16), dtype=torch.uint8)
  // cbg_cnt = N // 16 // 4  # Assuming N is divisible by 64
  // rb_cnt = M // 128  # Assuming M is divisible by 128
  // tmp = unswizzled_scale_factor_matrix.reshape(rb_cnt, 4, 32, cbg_cnt, 4)
  // tmp = torch.permute(tmp, (0, 3, 2, 1, 4))
  // swizzled_scale_factor_matrix = tmp.reshape((-1, 128, 4))

  constexpr uint32_t kTotalRowsPerBaseBlock = 128;
  constexpr uint32_t kRowsPerBaseBlockCol = 32;
  constexpr uint32_t kColsPerBaseBlockCol = 4;

  const size_t rb = row_idx / kTotalRowsPerBaseBlock;
  const size_t rem = row_idx % kTotalRowsPerBaseBlock;
  const size_t d4 = rem / kRowsPerBaseBlockCol;
  const size_t d3 = rem % kRowsPerBaseBlockCol;
  const size_t cbg = col_idx / kColsPerBaseBlockCol;
  const size_t d5 = col_idx % kColsPerBaseBlockCol;

  const size_t cbg_cnt = RoundUpDivide(col_length, kColsPerBaseBlockCol);
  // row-major offset in the logical shape
  // (rb_cnt , cbg_cnt , 32 , 4 , 4)
  // Magic number 16 below comes from the fact we have kColsPerBaseBlockCol = 4, and d4 ([0-128] /
  // 32 = [0-4])
  return ((rb * cbg_cnt + cbg) * kRowsPerBaseBlockCol + d3) * 16 + d4 * kColsPerBaseBlockCol + d5;
}

template <bool kReturnIdentity, bool kReturnTranspose, bool kIsE8Scaling, bool kAligned,
          typename CType, typename IType, typename OType, typename ScaleType, bool kSwizzledScale>
__global__ void __launch_bounds__(kThreadsPerBlock) block_scaled_1d_cast_transpose_kernel(
    const IType* const input, const float* global_amax, OType* const output_c,
    OType* const output_t, ScaleType* const tile_scales_inv_c, ScaleType* const tile_scales_inv_t,
    const size_t row_length, const size_t num_rows, const size_t scale_stride_x,
    const size_t scale_stride_y, const size_t scale_t_stride_x, const size_t scale_t_stride_y,
    const size_t kScaleBlockDim, const float epsilon) {
  using SMemVec = Vec<IType, kNVecSMem>;
  using OVec = Vec<OType, kNVecContainer>;
  union IVec {
    Vec<IType, kNVecIn> input_type;
    Vec<SMemVec, kNVecIn / kNVecSMem> smem_type;
  };

  extern __shared__ char smem_base[];
  SMemVec* smem = reinterpret_cast<SMemVec*>(&smem_base[0]);

  // Step 1: Load input to shared memory
  {
    constexpr int r_stride = kThreadsPerBlock / kNumThreadsLoad;  // stride in rows of shared memory
    constexpr int num_iterations = kTileDim / r_stride;
    const int c_s =
        (threadIdx.x % kNumThreadsLoad) * (kNVecIn / kNVecSMem);         // Column in shared memory
    int r_s = threadIdx.x / kNumThreadsLoad;                             // Row in shared memory
    const size_t c_g = (size_t)blockIdx.x * kTileDim + c_s * kNVecSMem;  // Column in global memory
    size_t r_g = (size_t)blockIdx.y * kTileDim + r_s;                    // Row in global memory
    const size_t stride_g = (size_t)r_stride * row_length;               // Stride in global memory
    const size_t num_ele =
        c_g < row_length ? min((size_t)kNVecIn, row_length - c_g) : 0;  // For not aligned case
    const IType* input_g = &input[r_g * row_length + c_g];  // Input address in global memory
#pragma unroll
    for (int iter = 0; iter < num_iterations; ++iter) {
      IVec input_vec;
      // Step 1.1: Load from global memory (input) to registers
      if constexpr (kAligned) {
        input_vec.input_type.VecLoadFrom(input_g);
      } else {
        if (r_g < num_rows) {
          input_vec.input_type.EleLoadFromIfNeeded(input_g, 0, num_ele);
        } else {
          input_vec.input_type.clear();
        }
      }
      // Step 1.2: Write to shared memory
#pragma unroll
      for (int i = 0; i < kNVecIn / kNVecSMem; ++i) {
        int c = c_s + i;
        int r = r_s;
        smem[r * kSMemCol + c] = input_vec.smem_type.data.ele[i];
      }
      // Step 1.3: Update input address, row index of shared memory, (and row index of global memory
      // for not aligned case)
      input_g += stride_g;
      r_s += r_stride;
      if constexpr (!kAligned) {
        r_g += r_stride;
      }
    }
  }

  __syncthreads();

  const int kNumThreadsReduce = kScaleBlockDim / kNVecOut;
  const float global_encode_scale =
      kIsE8Scaling ? 1.0f : ComputeGlobalEncodeScaleFP4(global_amax[0]);
  // Step 2: Cast and store to output_c
  if constexpr (kReturnIdentity) {
    constexpr int r_stride =
        kThreadsPerBlock / kNumThreadsStore;  // stride in rows of shared memory
    constexpr int num_iterations = kTileDim / r_stride;
    const int c_s =
        (threadIdx.x % kNumThreadsStore) * (kNVecOut / kNVecSMem);       // Column in shared memory
    int r_s = threadIdx.x / kNumThreadsStore;                            // Row in shared memory
    const size_t c_g = (size_t)blockIdx.x * kTileDim + c_s * kNVecSMem;  // Column in global memory
    size_t r_g = (size_t)blockIdx.y * kTileDim + r_s;                    // Row in global memory
    const size_t stride_g = (size_t)r_stride * row_length;               // Stride in global memory
    const size_t num_ele = c_g < row_length ? min((size_t)kNVecOut / kNFP4PerContainer,
                                                  (row_length - c_g) / kNFP4PerContainer)
                                            : 0;  // For not aligned case
    OType* output_g =
        &output_c[(r_g * row_length + c_g) / kNFP4PerContainer];  // Output address in global memory
    // Each kNumThreadsStore threads form a warp process one row, we need to find the lane id of
    // the first thread to do the reduction.
    const unsigned src_lane =
        (threadIdx.x % kThreadsPerWarp) / kNumThreadsReduce * kNumThreadsReduce;
    // This mask represents which threads should do the reduction together.
    const unsigned mask = ((1 << kNumThreadsReduce) - 1) << src_lane;
    const bool is_src_lane = (threadIdx.x % kNumThreadsReduce) == 0;
#pragma unroll
    for (int iter = 0; iter < num_iterations; ++iter) {
      SMemVec smem_vec[kNVecOut / kNVecSMem];
      // Step 2.1: Load from shared memory to registers
#pragma unroll
      for (int i = 0; i < kNVecOut / kNVecSMem; ++i) {
        int c = c_s + i;
        int r = r_s;
        smem_vec[i] = smem[r * kSMemCol + c];
      }
      // Step 2.2: Compute local amax
      CType amax = 0;
#pragma unroll
      for (int i = 0; i < kNVecOut / kNVecSMem; ++i) {
#pragma unroll
        for (int j = 0; j < kNVecSMem; ++j) {
          __builtin_assume(amax >= 0);
          amax = fmaxf(amax, fabsf(smem_vec[i].data.ele[j]));
        }
      }
      // Step 2.3: Reduce amax
      if constexpr (kIsE8Scaling) {
#pragma unroll
        for (int delta = kNumThreadsReduce / 2; delta > 0; delta /= 2) {
          const float other_amax = __shfl_down_sync(mask, amax, delta);
          __builtin_assume(amax >= 0);
          __builtin_assume(other_amax >= 0);
          amax = fmaxf(amax, other_amax);
        }
        amax = __shfl_sync(mask, amax, src_lane);
      }
      // Step 2.4: Compute scale
      ScaleType scale_inv =
          ComputeDecodeScaleFP4<OType, ScaleType, kIsE8Scaling>(amax, global_encode_scale);
      float encode_scale =
          ComputeEncodeScaleFP4<ScaleType, kIsE8Scaling>(scale_inv, global_encode_scale);
      // Step 2.5: Write scale_inv
      bool write_scale_inv = is_src_lane;
      if constexpr (!kAligned) {
        write_scale_inv &= (r_g < num_rows);
        write_scale_inv &= (c_g < row_length);
      }
      if (write_scale_inv) {
        size_t row_idx = (size_t)blockIdx.y * kTileDim + r_s;
        size_t col_idx = (size_t)blockIdx.x * (kNumThreadsStore / kNumThreadsReduce) +
                         ((size_t)threadIdx.x % kNumThreadsStore) / kNumThreadsReduce;
        if constexpr (kSwizzledScale) {
          size_t offset = scale_factor_swizzled_offset<ScaleType>(
              row_idx, col_idx, RoundUpDivide(row_length, kScaleBlockDim));
          tile_scales_inv_c[offset] = scale_inv;
        } else {
          tile_scales_inv_c[row_idx * scale_stride_y + col_idx * scale_stride_x] = scale_inv;
        }
      }
      // Step 2.6: Quantize
      OVec output_vec;
#pragma unroll
      for (int i = 0; i < kNVecOut / kNVecSMem; ++i) {
#pragma unroll
        for (int j = 0; j < kNVecSMem; j += kNFP4PerContainer) {
          // Pack two elements into __hip_bfloat162
          float2 f2;
          f2.x = ComputeOutputFP4<IType, ScaleType, kIsE8Scaling>(smem_vec[i].data.ele[j],
                                                                  encode_scale);
          f2.y = ComputeOutputFP4<IType, ScaleType, kIsE8Scaling>(smem_vec[i].data.ele[j + 1],
                                                                  encode_scale);
          // Convert to __hip_fp4x2_e2m1
          output_vec.data.ele[i] = __hip_cvt_float2_to_fp4x2(f2, __HIP_E2M1, hipRoundNearest);
        }
      }
      // Step 2.7: Store output_c
      if constexpr (kAligned) {
        output_vec.VecStoreTo(output_g);
      } else {
        if (r_g < num_rows) {
          output_vec.EleStoreToIfNeeded(output_g, 0, num_ele);
        }
      }
      // Step 2.8: Update output address, row index of shared memory (and row index of global memory
      // for not aligned case)
      output_g += stride_g / kNFP4PerContainer;
      r_s += r_stride;
      if constexpr (!kAligned) {
        r_g += r_stride;
      }
    }
  }

  // Step 3: Transpose, cast and store to output_t
  if constexpr (kReturnTranspose) {
    constexpr int c_stride =
        kThreadsPerBlock / kNumThreadsStore;  // Stride in columns of shared memory
    constexpr int num_iterations = kTileDim / (c_stride * kNVecSMem);
    const int r_s = (threadIdx.x % kNumThreadsStore) * kNVecOut;      // Row in shared memory
    int c_s = threadIdx.x / kNumThreadsStore;                         // Column in shared memory
    size_t r_g = (size_t)blockIdx.x * kTileDim + c_s * kNVecSMem;     // Row in global memory
    const size_t c_g = (size_t)blockIdx.y * kTileDim + r_s;           // Column in global memory
    const size_t stride_g = (size_t)c_stride * kNVecSMem * num_rows;  // Stride in global memory
    const size_t num_ele = c_g < num_rows ? min((size_t)kNVecOut / kNFP4PerContainer,
                                                (num_rows - c_g) / kNFP4PerContainer)
                                          : 0;  // For not aligned case
    OType* output_g =
        &output_t[(r_g * num_rows + c_g) / kNFP4PerContainer];  // Output address in global memory
    // Each kNumThreadsStore threads form a warp process one row, we need to find the lane id of
    // the first thread to do the reduction.
    const unsigned src_lane =
        (threadIdx.x % kThreadsPerWarp) / kNumThreadsReduce * kNumThreadsReduce;
    // This mask represents which threads should do the reduction together.
    const unsigned mask = ((1 << kNumThreadsReduce) - 1) << src_lane;
    const bool is_src_lane = (threadIdx.x % kNumThreadsReduce) == 0;
#pragma unroll
    for (int iter = 0; iter < num_iterations; ++iter) {
      SMemVec smem_vec[kNVecOut];
      // Step 3.1: Load from shared memory to registers
#pragma unroll
      for (int i = 0; i < kNVecOut; ++i) {
        int r = r_s + i;
        int c = c_s;
        smem_vec[i] = smem[r * kSMemCol + c];
      }
#pragma unroll
      for (int smem_idx = 0; smem_idx < kNVecSMem; ++smem_idx) {
        // Step 3.2: Compute local amax
        CType amax = 0;
#pragma unroll
        for (int i = 0; i < kNVecOut; ++i) {
          amax = fmaxf(amax, fabsf(smem_vec[i].data.ele[smem_idx]));
        }
        // Step 3.3: Reduce amax
        if constexpr (kIsE8Scaling) {
#pragma unroll
          for (int delta = kNumThreadsReduce / 2; delta > 0; delta /= 2) {
            const float other_amax = __shfl_down_sync(mask, amax, delta);
            __builtin_assume(amax >= 0);
            __builtin_assume(other_amax >= 0);
            amax = fmaxf(amax, other_amax);
          }
          amax = __shfl_sync(mask, amax, src_lane);
        }
        // Step 3.4: Compute scale
        ScaleType scale_inv =
            ComputeDecodeScaleFP4<OType, ScaleType, kIsE8Scaling>(amax, global_encode_scale);
        float encode_scale =
            ComputeEncodeScaleFP4<ScaleType, kIsE8Scaling>(scale_inv, global_encode_scale);
        // Step 3.5: Write scale_inv_t
        bool write_scale_inv = is_src_lane;
        if constexpr (!kAligned) {
          write_scale_inv &= (r_g + smem_idx < row_length);
          write_scale_inv &= (c_g < num_rows);
        }
        if (write_scale_inv) {
          size_t row_idx = (size_t)blockIdx.x * kTileDim + c_s * kNVecSMem + smem_idx;
          size_t col_idx = (size_t)blockIdx.y * (kNumThreadsStore / kNumThreadsReduce) +
                           ((size_t)threadIdx.x % kNumThreadsStore) / kNumThreadsReduce;
          if constexpr (kSwizzledScale) {
            size_t offset = scale_factor_swizzled_offset<ScaleType>(
                row_idx, col_idx, RoundUpDivide(num_rows, kScaleBlockDim));
            tile_scales_inv_t[offset] = scale_inv;
          } else {
            tile_scales_inv_t[row_idx * scale_t_stride_y + col_idx * scale_t_stride_x] = scale_inv;
          }
        }
        // Step 3.6: Quantize
        OVec output_vec;
#pragma unroll
        for (int i = 0; i < kNVecOut / kNFP4PerContainer; i += 1) {
          // Pack two elements into __hip_bfloat162
          float2 f2;
          f2.x = ComputeOutputFP4<IType, ScaleType, kIsE8Scaling>(
              smem_vec[2 * i].data.ele[smem_idx], encode_scale);
          f2.y = ComputeOutputFP4<IType, ScaleType, kIsE8Scaling>(
              smem_vec[2 * i + 1].data.ele[smem_idx], encode_scale);
          // Convert to __hip_fp4x2_e2m1
          output_vec.data.ele[i] = __hip_cvt_float2_to_fp4x2(f2, __HIP_E2M1, hipRoundNearest);
        }
        // Step 3.7: Store output_t
        if constexpr (kAligned) {
          output_vec.VecStoreTo(output_g + smem_idx * num_rows / kNFP4PerContainer);
        } else {
          if (r_g + smem_idx < row_length) {
            output_vec.EleStoreToIfNeeded(output_g + smem_idx * num_rows / kNFP4PerContainer, 0,
                                          num_ele);
          }
        }
      }
      // Step 3.8: Update output address, column index of shared memory (and row index of global
      // memory for not aligned case)
      output_g += stride_g / kNFP4PerContainer;
      c_s += c_stride;
      if constexpr (!kAligned) {
        r_g += c_stride * kNVecSMem;
      }
    }
  }
}

#endif  // if CUDA_VERSION >= 12080

}  // namespace
}  // namespace quantize_transpose_nvfp4

namespace transformer_engine::detail {

void quantize_transpose_vector_blockwise_fp4(const SimpleTensor& input,
                                             const SimpleTensor& global_amax,
                                             SimpleTensor& scale_inv, SimpleTensor& scale_inv_t,
                                             SimpleTensor& output, SimpleTensor& output_t,
                                             const float epsilon, const bool return_identity,
                                             const bool return_transpose, const bool pow2_scale,
                                             const bool swizzled_scale, hipStream_t stream) {
  // #if CUDA_VERSION >= 12080

  NVTE_API_CALL(quantize_transpose_vector_blockwise_fp4);

  // pow 2 scale is for MXFP4 since it's using E8M0 scaling
  // raise error if pow2_scale is true
  NVTE_CHECK(!pow2_scale, "No support for pow2_scale for MXFP4 for now");

  if (!return_identity && !return_transpose) {
    return;
  }

  const size_t row_length = input.shape[1];
  const size_t num_rows = input.shape[0];

  size_t scale_stride_x = 0;
  size_t scale_stride_y = 0;

  if (return_identity) {
    scale_stride_x = 1;
    scale_stride_y = scale_inv.shape[1];
  }

  size_t scale_t_stride_x = 0;
  size_t scale_t_stride_y = 0;

  if (return_transpose) {
    scale_t_stride_x = 1;
    scale_t_stride_y = scale_inv_t.shape[1];
  }

  using namespace quantize_transpose_nvfp4;

  const size_t num_blocks_x = RoundUpDivide(row_length, (size_t)kTileDim);
  const size_t num_blocks_y = RoundUpDivide(num_rows, (size_t)kTileDim);

  TRANSFORMER_ENGINE_TYPE_SWITCH_INPUT(
      input.dtype, InputType,

      TRANSFORMER_ENGINE_TYPE_SWITCH_FP4x2_ONLY(
          output.dtype, 2, OutputType,

          dim3 grid(num_blocks_x, num_blocks_y, 1);

          using ScaleType = fp8e4m3; constexpr int kScaleBlockDim = 16;
          constexpr bool kPow2Scale = false;

          const bool full_tile = row_length % kTileDim == 0 && num_rows % kTileDim == 0;

          TRANSFORMER_ENGINE_SWITCH_CONDITION(
              return_identity, kReturnIdentity,

              TRANSFORMER_ENGINE_SWITCH_CONDITION(
                  return_transpose, kReturnTranspose,

                  TRANSFORMER_ENGINE_SWITCH_CONDITION(
                      full_tile, kAligned,

                      TRANSFORMER_ENGINE_SWITCH_CONDITION(
                          swizzled_scale, kSwizzledScale,

                          size_t smem_bytes = kSMemSize * sizeof(InputType);
                          if (smem_bytes >= 48 * 1024) {
                            hipError_t err = hipFuncSetAttribute(reinterpret_cast<const void*>(
                                &block_scaled_1d_cast_transpose_kernel<
                                    kReturnIdentity), kReturnTranspose, kPow2Scale, kAligned, float,
                                    InputType, OutputType, ScaleType, kSwizzledScale>,
                                hipFuncAttributeMaxDynamicSharedMemorySize, smem_bytes);
                            NVTE_CHECK(err == hipSuccess,
                                       "Failed to set dynamic shared memory size.");
                          } block_scaled_1d_cast_transpose_kernel<kReturnIdentity, kReturnTranspose,
                                                                  kPow2Scale, kAligned, float,
                                                                  InputType, OutputType, ScaleType,
                                                                  kSwizzledScale>
                          <<<grid, kThreadsPerBlock, smem_bytes, stream>>>(
                              reinterpret_cast<const InputType*>(input.dptr),
                              reinterpret_cast<const float*>(global_amax.dptr),
                              reinterpret_cast<OutputType*>(output.dptr),
                              reinterpret_cast<OutputType*>(output_t.dptr),
                              reinterpret_cast<ScaleType*>(scale_inv.dptr),
                              reinterpret_cast<ScaleType*>(scale_inv_t.dptr), row_length, num_rows,
                              scale_stride_x, scale_stride_y, scale_t_stride_x, scale_t_stride_y,
                              kScaleBlockDim,
                              epsilon);)  // kSwizzledScale
                      )                   // kAligned
                  )                       // kReturnTranspose
              )                           // kReturnIdentity
          )                               // OutputType
      )                                   // InputType

  NVTE_CHECK_CUDA(hipGetLastError());

  // #else
  //   NVTE_CHECK(false, "Quantize vector blockwise fp4 is not supported for CUDA version < 12.8");
  // #endif // if CUDA_VERSION >= 12080
}

}  // namespace transformer_engine::detail
