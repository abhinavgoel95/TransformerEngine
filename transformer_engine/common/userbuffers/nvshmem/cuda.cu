#include "hip/hip_runtime.h"
#include "te_nvshmem.h"

#include <nvshmem.h>

#include <memory>
#include <cstdio>
#include <hipblas.h>
#include <iostream>
#include <cstdint>
#include <hip/hip_fp8.h>
#include <hip/hip_fp16.h>

#include "macros.hpp.inc"

// stream_t

stream_t::stream_t() {
    CUDA_CHECK(hipStreamCreate(&stream));
    alive = true;
}

stream_t::~stream_t() {
    if(alive) {
        CUDA_CHECK(hipStreamDestroy(stream));
    }
}

void stream_t::synchronize() const {
    ASSERT(alive);
    CUDA_CHECK(hipStreamSynchronize(stream));
}

stream_t::stream_t(stream_t&& that) {
    stream = that.stream;
    alive = that.alive;
    that.stream = (hipStream_t)nullptr;
    that.alive = false;
}

stream_t& stream_t::operator=(stream_t&& that) {
    std::swap(stream, that.stream);
    std::swap(alive, that.alive);
    return *this;
}

stream_t::operator hipStream_t() const { 
    ASSERT(alive);
    return stream;
}

hipStream_t stream_t::handle() const { 
    ASSERT(alive);
    return stream;
}

void stream_t::wait(hipEvent_t event) const {
    ASSERT(alive);
    CUDA_CHECK(hipStreamWaitEvent(stream, event));
}

// event_t

event_t::event_t() {
    CUDA_CHECK(hipEventCreate(&event));
    alive = true;
}

event_t::~event_t() {
    if(alive) {
        CUDA_CHECK(hipEventDestroy(event));
    }
}

event_t::event_t(event_t&& that) {
    event = that.event;
    alive = that.alive;
    that.event = (hipEvent_t)nullptr;
    that.alive = false;
}

event_t& event_t::operator=(event_t&& that) {
    std::swap(event, that.event);
    std::swap(alive, that.alive);
    return *this;
}

event_t::operator hipEvent_t() const { 
    ASSERT(alive);
    return event;
}

hipEvent_t event_t::handle() const { 
    ASSERT(alive);
    return event;
}

void event_t::record(hipStream_t stream) const {
    ASSERT(alive);
    CUDA_CHECK(hipEventRecord(event, stream));
}

float event_t::elapsed_time_ms(hipEvent_t stop) const {
    ASSERT(alive);
    float time_ms = 0;
    CUDA_CHECK(hipEventElapsedTime(&time_ms, event, stop));
    return time_ms;
}

// nvshmem_vector_t

template<typename T> 
nvshmem_vector_t<T>::nvshmem_vector_t(size_t size) : device_vector_view_t<T>(nullptr, size) {
    if (this->_size == 0) {
        this->_ptr_d = nullptr;
    } else {
        this->_ptr_d = (T*)nvshmem_malloc(sizeof(T) * size);
        ASSERT(this->_ptr_d != nullptr);
    }
}

template<typename T> 
nvshmem_vector_t<T>::nvshmem_vector_t(const std::vector<T>& data) : device_vector_view_t<T>(nullptr, data.size()) {
    if (data.size() == 0) {
        this->_ptr_d = nullptr;
    } else {
        this->_ptr_d = (T*)nvshmem_malloc(sizeof(T) * data.size());
        ASSERT(this->_ptr_d != nullptr);
        CUDA_CHECK(hipMemcpy(this->_ptr_d, data.data(), data.size() * sizeof(T), hipMemcpyDefault));
    }
}

template<typename T> 
nvshmem_vector_t<T>::~nvshmem_vector_t() {
    if(this->_ptr_d != nullptr) {
        nvshmem_free(this->_ptr_d);
    }
}

// device_vector_const_view_t

template<typename T>
device_vector_const_view_t<T>::device_vector_const_view_t(device_vector_const_view_t<T>&& that) {
    this->_size = that._size;
    this->_ptr_d = that._ptr_d;
    that._size = 0;
    that._ptr_d = nullptr;
}

template<typename T>
device_vector_const_view_t<T>& device_vector_const_view_t<T>::operator=(device_vector_const_view_t<T>&& that) {
    std::swap(this->_size, that._size);
    std::swap(this->_ptr_d, that._ptr_d);
    return *this;
}

template<typename T>
device_vector_const_view_t<T>::operator std::vector<T>() {
    std::vector<T> out(_size);
    CUDA_CHECK(hipMemcpy(out.data(), this->_ptr_d, this->_size * sizeof(T), hipMemcpyDefault));
    return out;
}

// device_vector_view_t

template<typename T>
device_vector_view_t<T>::device_vector_view_t(device_vector_view_t<T>&& that) {
    this->_size = that._size;
    this->_ptr_d = that._ptr_d;
    that._size = 0;
    that._ptr_d = nullptr;
}

template<typename T>
device_vector_view_t<T>& device_vector_view_t<T>::operator=(device_vector_view_t<T>&& that) {
    std::swap(this->_size, that._size);
    std::swap(this->_ptr_d, that._ptr_d);
    return *this;
}

template<typename T>
device_vector_view_t<T>::operator std::vector<T>() {
    return (std::vector<T>) device_vector_const_view_t(this->_ptr_d, this->_size);
}

// device_vector_t

template<typename T>
device_vector_t<T>::device_vector_t(size_t size) : device_vector_view_t<T>(nullptr, size) {
    if (size == 0) {
        this->_ptr_d = nullptr;
    } else {
        CUDA_CHECK(hipMalloc(&this->_ptr_d, size * sizeof(T)));
        CUDA_CHECK(hipMemset(this->_ptr_d, 0, size * sizeof(T)));
    }
}

template<typename T>
device_vector_t<T>::device_vector_t(const std::vector<T>& data) : device_vector_view_t<T>(nullptr, data.size()) {
    if (data.size() == 0) {
        this->_ptr_d = nullptr;
    } else {
        CUDA_CHECK(hipMalloc(&this->_ptr_d, data.size() * sizeof(T)));
        CUDA_CHECK(hipMemcpy(this->_ptr_d, data.data(), data.size() * sizeof(T), hipMemcpyDefault));
    }
}

template<typename T>
device_vector_t<T>::~device_vector_t() {
    if (this->_ptr_d != nullptr) {
        CUDA_CHECK(hipFree(this->_ptr_d));
    }
}

//////// TODO: fix this madness

template nvshmem_vector_t<hip_bfloat16>::nvshmem_vector_t(size_t size);
template nvshmem_vector_t<hip_bfloat16>::nvshmem_vector_t(const std::vector<hip_bfloat16>& data);
template nvshmem_vector_t<hip_bfloat16>::~nvshmem_vector_t();

template nvshmem_vector_t<uint64_t>::nvshmem_vector_t(size_t size);
template nvshmem_vector_t<uint64_t>::nvshmem_vector_t(const std::vector<uint64_t>& data);
template nvshmem_vector_t<uint64_t>::~nvshmem_vector_t();

template nvshmem_vector_t<char>::nvshmem_vector_t(size_t size);
template nvshmem_vector_t<char>::nvshmem_vector_t(const std::vector<char>& data);
template nvshmem_vector_t<char>::~nvshmem_vector_t();

template device_vector_const_view_t<hip_bfloat16>::device_vector_const_view_t(device_vector_const_view_t<hip_bfloat16>&&);
template device_vector_const_view_t<hip_bfloat16>& device_vector_const_view_t<hip_bfloat16>::operator=(device_vector_const_view_t<hip_bfloat16>&&);
template device_vector_const_view_t<hip_bfloat16>::operator std::vector<hip_bfloat16>();

template device_vector_const_view_t<__hip_fp8_e4m3_fnuz>::device_vector_const_view_t(device_vector_const_view_t<__hip_fp8_e4m3_fnuz>&&);
template device_vector_const_view_t<__hip_fp8_e4m3_fnuz>& device_vector_const_view_t<__hip_fp8_e4m3_fnuz>::operator=(device_vector_const_view_t<__hip_fp8_e4m3_fnuz>&&);
template device_vector_const_view_t<__hip_fp8_e4m3_fnuz>::operator std::vector<__hip_fp8_e4m3_fnuz>();

template device_vector_const_view_t<__hip_fp8_e5m2_fnuz>::device_vector_const_view_t(device_vector_const_view_t<__hip_fp8_e5m2_fnuz>&&);
template device_vector_const_view_t<__hip_fp8_e5m2_fnuz>& device_vector_const_view_t<__hip_fp8_e5m2_fnuz>::operator=(device_vector_const_view_t<__hip_fp8_e5m2_fnuz>&&);
template device_vector_const_view_t<__hip_fp8_e5m2_fnuz>::operator std::vector<__hip_fp8_e5m2_fnuz>();

template device_vector_const_view_t<char>::device_vector_const_view_t(device_vector_const_view_t<char>&&);
template device_vector_const_view_t<char>& device_vector_const_view_t<char>::operator=(device_vector_const_view_t<char>&&);
template device_vector_const_view_t<char>::operator std::vector<char>();

template device_vector_const_view_t<uint64_t>::device_vector_const_view_t(device_vector_const_view_t<uint64_t>&&);
template device_vector_const_view_t<uint64_t>& device_vector_const_view_t<uint64_t>::operator=(device_vector_const_view_t<uint64_t>&&);
template device_vector_const_view_t<uint64_t>::operator std::vector<uint64_t>();

template device_vector_view_t<hip_bfloat16>::operator std::vector<hip_bfloat16>();
template device_vector_view_t<hip_bfloat16>::device_vector_view_t(device_vector_view_t<hip_bfloat16>&&);
template device_vector_view_t<hip_bfloat16>& device_vector_view_t<hip_bfloat16>::operator=(device_vector_view_t<hip_bfloat16>&&);

template device_vector_view_t<char>::operator std::vector<char>();
template device_vector_view_t<char>::device_vector_view_t(device_vector_view_t<char>&&);
template device_vector_view_t<char>& device_vector_view_t<char>::operator=(device_vector_view_t<char>&&);

template device_vector_view_t<int32_t>::operator std::vector<int32_t>();
template device_vector_view_t<int32_t>::device_vector_view_t(device_vector_view_t<int32_t>&&);
template device_vector_view_t<int32_t>& device_vector_view_t<int32_t>::operator=(device_vector_view_t<int32_t>&&);

template device_vector_view_t<uint64_t>::operator std::vector<uint64_t>();
template device_vector_view_t<uint64_t>::device_vector_view_t(device_vector_view_t<uint64_t>&&);
template device_vector_view_t<uint64_t>& device_vector_view_t<uint64_t>::operator=(device_vector_view_t<uint64_t>&&);

template device_vector_t<hip_bfloat16>::device_vector_t(size_t size);
template device_vector_t<hip_bfloat16>::device_vector_t(const std::vector<hip_bfloat16>& data);
template device_vector_t<hip_bfloat16>::~device_vector_t();

template device_vector_t<int32_t>::device_vector_t(size_t size);
template device_vector_t<int32_t>::device_vector_t(const std::vector<int32_t>& data);
template device_vector_t<int32_t>::~device_vector_t();

template device_vector_t<char>::device_vector_t(size_t size);
template device_vector_t<char>::device_vector_t(const std::vector<char>& data);
template device_vector_t<char>::~device_vector_t();

template device_vector_t<__hip_fp8_e4m3_fnuz>::device_vector_t(size_t size);
template device_vector_t<__hip_fp8_e4m3_fnuz>::device_vector_t(const std::vector<__hip_fp8_e4m3_fnuz>& data);
template device_vector_t<__hip_fp8_e4m3_fnuz>::~device_vector_t();

template device_vector_t<__hip_fp8_e5m2_fnuz>::device_vector_t(size_t size);
template device_vector_t<__hip_fp8_e5m2_fnuz>::device_vector_t(const std::vector<__hip_fp8_e5m2_fnuz>& data);
template device_vector_t<__hip_fp8_e5m2_fnuz>::~device_vector_t();

