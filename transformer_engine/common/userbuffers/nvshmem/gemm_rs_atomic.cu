#include "hip/hip_runtime.h"
#include "macros.hpp.inc"

#include <cxxopts.hpp>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <mpi.h>
#include <nccl.h>
#include <hipblas.h>
#include <iostream>

#include "helpers.hpp"
#include "mpi_helpers.hpp"
#include "nccl_helpers.hpp"
#include "timings_helpers.hpp"
#include "cublas_helpers.hpp"

#include "te_nvshmem.h"
#include "gemm_rs.hpp"

using TA = __hip_fp8_e4m3_fnuz;
using TB = __hip_fp8_e4m3_fnuz;
using TC = __hip_bfloat16;

int main(int argc, char** argv) {
    return gemm_rs_main<TA, TB, TC, cublasmp_gemm_rs_atomic_t<TA, TB, TC>>("GEMM+RS(atomic). ", argc, argv);
}
