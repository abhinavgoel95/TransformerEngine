#include "hip/hip_runtime.h"
#include <nvshmem.h>
#include <cstdio>
#include <hip/hip_bf16.h>
#include <string>
#include <cstdlib>

#include "te_nvshmem.h"

#include "macros.hpp.inc"

static const bool TE_NVSHMEM_DEBUG = (std::getenv("TE_NVSHMEM_DEBUG") != nullptr && std::string(std::getenv("TE_NVSHMEM_DEBUG")) == "1");

nvshmem_comm_t::nvshmem_comm_t(int my_pe, int n_pes) : 
    my_pe(my_pe), n_pes(n_pes) {};

std::unique_ptr<nvshmem_comm_t> nvshmem_comm_t::create(int my_rank, int num_ranks) {
    const int my_pe = nvshmem_my_pe();
    const int n_pes = nvshmem_n_pes();
    if(TE_NVSHMEM_DEBUG) {
        printf("NVSHMEM initialized, PE %d/%d\n", my_pe, n_pes);
    }
    ASSERT_EQ(my_rank, my_pe);
    ASSERT_EQ(n_pes, num_ranks);
    ASSERT(my_pe >= 0);
    ASSERT(n_pes > 0);
    return std::unique_ptr<nvshmem_comm_t>(new nvshmem_comm_t(my_rank, num_ranks));
}

nvshmem_comm_t::~nvshmem_comm_t() {
    nvshmem_finalize();
}

void* nvshmem_comm_t::malloc(size_t size) {
    if(size == 0) {
        size = 1;
    }
    void* ptr = nvshmem_malloc(size);
    ASSERT(ptr != nullptr);
    return ptr;
}

void nvshmem_comm_t::free(void* ptr) {
    nvshmem_free(ptr);
}

nvshmem_comm_t::error_t nvshmem_comm_t::barrier_all() {
    nvshmem_barrier_all();
    return nvshmem_comm_t::error_t::SUCCESS; 
}

nvshmem_comm_t::error_t nvshmem_comm_t::sync_all_on_stream(hipStream_t stream) {
    nvshmemx_sync_all_on_stream(stream);
    return nvshmem_comm_t::error_t::SUCCESS; 
}

nvshmem_comm_t::error_t nvshmem_comm_t::barrier_all_on_stream(hipStream_t stream) {
    nvshmemx_barrier_all_on_stream(stream);
    return nvshmem_comm_t::error_t::SUCCESS; 
}

int nvshmem_comm_t::this_pe() const {
    return my_pe;
}

int nvshmem_comm_t::num_pes() const {
    return n_pes;
}

template<> 
__device__ __forceinline__ int4 nvshmem_comm_t::nvshmem_g<int4>(const int4* ptr, int pe) {
    static_assert(sizeof(int4) == 2 * sizeof(uint64_t));
    int4 v;
    uint64_t* p = (uint64_t*)(&v);
    p[0] = nvshmem_uint64_g(((uint64_t*)ptr) + 0, pe);
    p[1] = nvshmem_uint64_g(((uint64_t*)ptr) + 1, pe);
    return v;
}

__global__ void set_kernel(int *flag, int value) {
    *flag = value;
    asm volatile("fence.sc.gpu;\n");
}

nvshmem_comm_t::error_t nvshmem_comm_t::set(int* flag, int value, hipStream_t stream) {
    set_kernel<<<1, 1, 0, stream>>>(flag, value);
    CUDA_CHECK(hipGetLastError());
    return nvshmem_comm_t::error_t::SUCCESS;
}

// For producer consumer: wait_on_atomic_and_set_kernel(flag, signal=0, value=1)
__global__ void wait_on_atomic_and_set_kernel(int *flag, int signal, int value) {
    while (signal != (atomicCAS(flag, signal, signal))) {
        // spin
    }
    *flag = value;
    // fence, to ensure results are visible to following kernel
    asm volatile("fence.sc.gpu;\n");
}

nvshmem_comm_t::error_t nvshmem_comm_t::wait_on_atomic_and_set(int* flag, int signal, int value, hipStream_t stream) {
    if(TE_NVSHMEM_DEBUG) {
        printf("[%d] wait_on_atomic_and_set flag %p signal %d set %d stream %p\n", my_pe, flag, signal, value, (void*)stream);
    }
    wait_on_atomic_and_set_kernel<<<1, 1, 0, stream>>>(flag, signal, value);
    CUDA_CHECK(hipGetLastError());
    return nvshmem_comm_t::error_t::SUCCESS;
}

nvshmem_p2p_t::nvshmem_p2p_t(int my_pe, int n_pes, nvshmem_vector_t<uint64_t> flags) :
    nvshmem_comm_t(my_pe, n_pes), flags(std::move(flags)), counters(n_pes, 0) {};

std::unique_ptr<nvshmem_p2p_t> nvshmem_p2p_t::create(int my_rank, int num_ranks) {
    nvshmem_init();
    const int my_pe = nvshmem_my_pe();
    const int n_pes = nvshmem_n_pes();
    if(TE_NVSHMEM_DEBUG) {
        printf("NVSHMEM initialized, PE %d/%d\n", my_pe, n_pes);
    }
    ASSERT(my_pe == my_rank);
    ASSERT(num_ranks == n_pes);
    ASSERT(my_pe >= 0);
    ASSERT(n_pes > 0);
    nvshmem_vector_t<uint64_t> flags(num_ranks);
    return std::unique_ptr<nvshmem_p2p_t>(new nvshmem_p2p_t(my_pe, n_pes, std::move(flags)));
}

nvshmem_comm_t::error_t nvshmem_p2p_t::send_and_signal(const void* src, void* dst, size_t size, int peer, hipStream_t stream) {
    ASSERT(peer < this->n_pes);
    ASSERT(peer >= 0);
    ASSERT(this->flags.size() == (size_t)this->n_pes);
    // Push-send mode
    uint64_t* flag = this->flags.data() + my_pe;
    uint64_t  signal = 1;
    int       sig_op = NVSHMEM_SIGNAL_ADD;
    char*     ptr_dst = (char*)dst;
    const char* ptr_src = (const char*)src;
    if(TE_NVSHMEM_DEBUG) {
        printf("[%d] putmem %p -> %p (pe %d) (flag %p) stream %p\n", my_pe, ptr_src, ptr_dst, peer, flag, (void*)stream);
    }
    nvshmemx_putmem_signal_on_stream(ptr_dst, ptr_src, size, flag, signal, sig_op, peer, stream);
    return nvshmem_comm_t::error_t::SUCCESS;
}

nvshmem_comm_t::error_t nvshmem_p2p_t::wait(int peer, hipStream_t stream) {
    ASSERT(peer < this->n_pes);
    ASSERT(peer >= 0);
    ASSERT((size_t)peer < counters.size());
    ASSERT(this->flags.size() == (size_t)this->n_pes);
    // Push-send mode
    uint64_t* flag = this->flags.data() + peer;
    uint64_t  signal = (counters[peer] + 1);
    if(TE_NVSHMEM_DEBUG) {
        printf("[%d] wait until (flag %p) >= %d, stream %p\n", my_pe, flag, (int)signal, (void*)stream);
    }
    nvshmemx_uint64_wait_until_on_stream(flag, NVSHMEM_CMP_GE, signal, stream);
    counters[peer] += 1;
    return nvshmem_comm_t::error_t::SUCCESS;
}

template<typename T> 
nvshmem_vector_t<T> nvshmem_comm_t::make_vector(size_t size) {
    return nvshmem_vector_t<T>(size);
}

template<typename T> 
nvshmem_vector_t<T> nvshmem_comm_t::make_vector(const std::vector<T>& data) {
    return nvshmem_vector_t<T>(data);
}

////////

nvshmem_reduce_scatter_t::nvshmem_reduce_scatter_t(int my_pe, int n_pes, nvshmem_vector_t<uint64_t> flags) :
    nvshmem_comm_t(my_pe, n_pes), flags(std::move(flags)), counter(0) {};

std::unique_ptr<nvshmem_reduce_scatter_t> nvshmem_reduce_scatter_t::create(int my_rank, int num_ranks) {
    nvshmem_init();
    const int my_pe = nvshmem_my_pe();
    const int n_pes = nvshmem_n_pes();
    if(TE_NVSHMEM_DEBUG) {
        printf("NVSHMEM initialized, PE %d/%d\n", my_pe, n_pes);
    }
    ASSERT_EQ(my_pe, my_rank);
    ASSERT_EQ(num_ranks, n_pes);
    ASSERT(my_pe >= 0);
    ASSERT(n_pes > 0);
    nvshmem_vector_t<uint64_t> rs_flags(n_pes);
    return std::unique_ptr<nvshmem_reduce_scatter_t>(new nvshmem_reduce_scatter_t(my_pe, n_pes, std::move(rs_flags)));
}

// Reduce (add) the matrices and scatter the rows accross PEs + sync PEs. 
// This syncs PEs at the beginning of the kernel: this means kernels on different PEs will wait on each other before starting
// to read data from each other.
//
// Example with 2 PEs and 4x8 matrices
// 
// Input
// -----
// Inputs are row-major, with leading dimension src_ld, shape src_rows x src_cols
//
// On PE0:
// src = [  0  1  2  3  4  5  6  7]
//       [  8  9 10 11 12 13 14 15]
//       [ 16 17 18 19 20 21 22 23]
//       [ 24 25 26 27 28 29 30 31] 
// 
//
// On PE1:
// src = [ 32 33 34 35 36 37 38 39]
//       [ 40 41 42 43 44 45 46 47]
//       [ 48 49 50 51 52 53 54 55]
//       [ 56 57 58 59 60 61 62 63]
// 
// Output
// -----
// Outputs are row-major, with leading dimension dst_ld, shape dst_rows x dst_cols
//
// On PE0:
// dst = [32 34 36 38 40 42 44 46]
//       [48 50 52 54 56 58 60 62]
//
// On PE1:
// dst = [64 66 68 70 72 74 76 78]
//       [80 82 84 86 88 90 92 94]
//
template<typename T, typename F, int num_pes>
__global__ void reduce_scatter_kernel(const T* src, 
                                      size_t src_rows, 
                                      size_t src_cols, 
                                      size_t src_ld, 
                                      T* dst, 
                                      size_t dst_rows, 
                                      size_t dst_cols, 
                                      size_t dst_ld,
                                      uint64_t* flags,
                                      uint64_t signal) {

    const int my_pe = nvshmem_my_pe();
    const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Increment flags[my_pe] on remote PE dest_pe
    if(threadIdx.x < num_pes && blockIdx.x == 0) {
        const int dest_pe = threadIdx.x;
        uint64_t* flag = flags + my_pe; // == &flags[my_pe];
        // printf("[%d %d] Incrementing %p (%p + %d) on %d\n", blockIdx.x, threadIdx.x, flag, flags, my_pe, dest_pe);
        nvshmem_uint64_atomic_inc(flag, dest_pe);
    }
    // Wait for flags[pe] to be incremented, for all pe
    // (each thread waits for one pe)
    if(threadIdx.x < num_pes) {
        const int source_pe = threadIdx.x;
        uint64_t* flag = flags + source_pe; // == &flags[source_pe];
        // printf("[%d %d] Waiting on %p (%p + %d) to reach %d on %d\n", blockIdx.x, threadIdx.x, flag, flags, source_pe, (int)signal, my_pe);
        nvshmem_uint64_wait_until(flag, NVSHMEM_CMP_GE, signal);
    }
    __syncthreads();

    if(tid >= dst_rows * dst_cols) {
        return;
    }

    const size_t dst_col = tid % dst_cols;
    const size_t dst_row = tid / dst_cols;
    const size_t dst_mem_idx = dst_col + dst_row * dst_ld;

    T reduced = F::init();

    // rows
    const size_t src_col = dst_col; // cols: my_pe * dst_cols + dst_col;
    const size_t src_row = my_pe * dst_rows + dst_row; // cols: dst_row
    const size_t src_mem_idx = src_col + src_row * src_ld;

    #pragma unroll
    for(int peid = 0; peid < num_pes; peid++) {

        // Shuffle PEs
        int warp = blockIdx.x + (threadIdx.x >> 5);
        int pe = (peid + my_pe + warp) & (num_pes - 1);

        T v = nvshmem_comm_t::nvshmem_g<T>(src + src_mem_idx, pe);

        // if(threadIdx.x == 0 && blockIdx.x == 0) printf("[%d] loading %p (base %p, idx %d, pe %d, src %d %d, dst %d %d, tid %d, src(s) %d %d, dst(s) %d %d) %f %f %f %f %f %f %f %f\n", my_pe, src + src_mem_idx, src, 
        //     (int)src_mem_idx, pe, (int)src_row, (int)src_col, (int)dst_row, (int)dst_col, (int)tid, (int)src_rows, (int)src_cols, (int)dst_rows, (int)dst_cols,
        //     (float)((hip_bfloat16*)&v)[0],
        //     (float)((hip_bfloat16*)&v)[1],
        //     (float)((hip_bfloat16*)&v)[2],
        //     (float)((hip_bfloat16*)&v)[3],
        //     (float)((hip_bfloat16*)&v)[4],
        //     (float)((hip_bfloat16*)&v)[5],
        //     (float)((hip_bfloat16*)&v)[6],
        //     (float)((hip_bfloat16*)&v)[7]);

        reduced = F::reduce(reduced, v);

        // if(threadIdx.x == 0 && blockIdx.x == 0) printf("[%d] reduced %f %f %f %f %f %f %f %f\n", my_pe, 
        //     (float)((hip_bfloat16*)&reduced)[0],
        //     (float)((hip_bfloat16*)&reduced)[1],
        //     (float)((hip_bfloat16*)&reduced)[2],
        //     (float)((hip_bfloat16*)&reduced)[3],
        //     (float)((hip_bfloat16*)&reduced)[4],
        //     (float)((hip_bfloat16*)&reduced)[5],
        //     (float)((hip_bfloat16*)&reduced)[6],
        //     (float)((hip_bfloat16*)&reduced)[7]);
    }

    // if(threadIdx.x == 0 && blockIdx.x == 0) printf("[%d] storing %p (base %p, idx %d) %f %f %f %f %f %f %f %f\n", my_pe, &dst[dst_mem_idx], dst, (int)dst_mem_idx,
    //         (float)((hip_bfloat16*)&reduced)[0],
    //         (float)((hip_bfloat16*)&reduced)[1],
    //         (float)((hip_bfloat16*)&reduced)[2],
    //         (float)((hip_bfloat16*)&reduced)[3],
    //         (float)((hip_bfloat16*)&reduced)[4],
    //         (float)((hip_bfloat16*)&reduced)[5],
    //         (float)((hip_bfloat16*)&reduced)[6],
    //         (float)((hip_bfloat16*)&reduced)[7]);

    dst[dst_mem_idx] = reduced;

}

template<typename T>
struct adder_int4 {

    static_assert(sizeof(int4) % sizeof(T) == 0);

    __forceinline__ __device__ static int4 init() {
        int4 out;
        for(int i = 0; i < sizeof(int4) / sizeof(T); i++) {
            ((T*)(&out))[i] = 0;
        }
        return out;
    }

    __forceinline__ __device__ static int4 reduce(int4 lhs, int4 rhs) {
        int4 out;
        for(int i = 0; i < sizeof(int4) / sizeof(T); i++) {
            ((T*)(&out))[i] = ((T*)(&lhs))[i] + ((T*)(&rhs))[i];
        }
        return out;
    }

};

/**
 * Matrices are rows major
 * 
 * Inputs: on each GPU, a `src` of size src_rows x src_cols, row-major, leading dimension src_ld
 * 
 * Outputs: with nPEs GPUs, a `dst` of size (src_rows / nPEs) x src_cols, row-major, leading dimension dst_ld
 * 
 */
template<typename T> 
nvshmem_comm_t::error_t nvshmem_reduce_scatter_t::reduce_scatter(const T* src, size_t src_rows, size_t src_cols, size_t src_ld, T* dst, size_t dst_ld, hipStream_t stream) {

    if(TE_NVSHMEM_DEBUG) {
        printf("[%d] reduce_scatter %p (%zu %zu %zu) -> %p (%zu) |T| %zu, stream %p\n", my_pe, src, src_rows, src_cols, src_ld, dst, dst_ld, sizeof(T), (void*)stream);
    }

    const int npes = nvshmem_n_pes();

    using Tv = int4;
    constexpr unsigned vec_size = sizeof(Tv) / sizeof(T);

    // Check it's OKay to vectorize & that source lds are >= cols
    ASSERT(src_cols >= vec_size && src_cols % vec_size == 0);
    const size_t src_cols_v = src_cols / vec_size;
    const size_t src_rows_v = src_rows;
    ASSERT(src_ld >= src_cols && src_ld % vec_size == 0);
    const size_t src_ld_v = src_ld / vec_size;

    // Distribute cols
    // const size_t dst_cols_v = src_cols_v / npes;
    // const size_t dst_rows_v = src_rows_v;

    // ASSERT(src_cols % npes == 0);
    // const size_t dst_cols = src_cols / npes;
    // ASSERT(dst_ld >= dst_cols && dst_ld % vec_size == 0);
    // const size_t dst_ld_v = dst_ld / vec_size;

    // Distribute rows
    const size_t dst_cols_v = src_cols_v;
    ASSERT(src_rows_v % npes == 0);
    const size_t dst_rows_v = src_rows_v / npes;

    const size_t dst_cols = src_cols;
    ASSERT(dst_ld >= dst_cols && dst_ld % vec_size == 0);
    const size_t dst_ld_v = dst_ld / vec_size;

    // Launch
    const size_t grid_size = dst_rows_v * dst_cols_v;
    const size_t block_size = 128;
    const size_t num_blocks = (grid_size + block_size - 1) / block_size;
    ASSERT(block_size >= (size_t)npes);

    const Tv* src_v = (const Tv*) src;
          Tv* dst_v =       (Tv*) dst;

    ASSERT(this->flags.size() == (size_t)npes);
    uint64_t* flags = this->flags.data();
    uint64_t  signal = counter + 1;
    counter += 1;

    if(TE_NVSHMEM_DEBUG) {
        printf("[%d] reduce_scatter_kernel<<<%zu %zu>>> %p (%zu x %zu ld %zu) -> %p  (%zu x %zu ld %zu), |T| %zu, flags %p signal %zu\n", my_pe, num_blocks, block_size, src_v, src_rows_v, src_cols_v, src_ld_v, dst_v, dst_rows_v, dst_cols_v, dst_ld_v, sizeof(Tv), flags, signal);
    }

    switch(npes) {
        case 2:
            reduce_scatter_kernel<Tv, adder_int4<__hip_bfloat16>, 2> <<<num_blocks, block_size, 0, stream>>> (src_v, src_rows_v, src_cols_v, src_ld_v, dst_v, dst_rows_v, dst_cols_v, dst_ld_v, flags, signal);
            break;
        case 4:
            reduce_scatter_kernel<Tv, adder_int4<__hip_bfloat16>, 4> <<<num_blocks, block_size, 0, stream>>> (src_v, src_rows_v, src_cols_v, src_ld_v, dst_v, dst_rows_v, dst_cols_v, dst_ld_v, flags, signal);
            break;
        case 8:
            reduce_scatter_kernel<Tv, adder_int4<__hip_bfloat16>, 8> <<<num_blocks, block_size, 0, stream>>> (src_v, src_rows_v, src_cols_v, src_ld_v, dst_v, dst_rows_v, dst_cols_v, dst_ld_v, flags, signal);
            break;
        default:
            printf("Unsupported npes, got %d\n", npes);
            ASSERT(false);
    }
    
    CUDA_CHECK(hipGetLastError());

    return nvshmem_comm_t::error_t::SUCCESS;
}

///////

template nvshmem_vector_t<hip_bfloat16> nvshmem_comm_t::make_vector<hip_bfloat16>(size_t size);
template nvshmem_vector_t<hip_bfloat16> nvshmem_comm_t::make_vector<hip_bfloat16>(const std::vector<hip_bfloat16>& data);

template nvshmem_vector_t<char> nvshmem_comm_t::make_vector<char>(size_t size);
template nvshmem_vector_t<char> nvshmem_comm_t::make_vector<char>(const std::vector<char>& data);

template nvshmem_comm_t::error_t nvshmem_reduce_scatter_t::reduce_scatter<hip_bfloat16>(const hip_bfloat16* src, size_t rows, size_t cols, size_t src_ld, hip_bfloat16* dst, size_t dst_ld, hipStream_t stream);